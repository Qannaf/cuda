
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using namespace std;
__global__ void matrixMul(const int *a, const int *b, int *c, int N);
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c, int N);
void afficheMatrix(vector<int>& m,int line, int colone);


int main() 
{
  int N = 1 << 3;
  size_t bytes = N * N * sizeof(int);

  // CPU
  vector<int> h_a(N * N);
  vector<int> h_b(N * N);
  vector<int> h_c(N * N);
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // GPU
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // CPU --->  GPU
  hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

  
  int THREADS = 1 << 2;
  int BLOCKS = N / THREADS;
  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS, BLOCKS);

  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c, N);

  // CPU ---> GPU
  hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  
  verify_result(h_a, h_b, h_c, N);

  cout << "terminé avec succès"<<endl;

  afficheMatrix(h_a,N,N);
  afficheMatrix(h_b,N,N);
  afficheMatrix(h_c,N,N);


  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}



__global__ void matrixMul(const int *a, const int *b, int *c, int N) {
  // Compute each thread's global row and column index
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Iterate over row, and down column
  int tmp = 0;
  for (int k = 0; k < N; k++) {
    // Accumulate results for a single element
    tmp += a[row * N + k] * b[k * N + col];
  }

  // Write back the result
  c[row * N + col] = tmp;
}

// Check result on the CPU
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c, int N) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[i * N + k] * b[k * N + j];
      }

      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
  }
}


void afficheMatrix(vector<int>& m,int line, int colone)
{ 
  for (int i = 0; i <line;  i++) 
  {
    for (int j = 0; j < colone; j++)
    { 
      cout<<m[i]<<" ";
    }
  cout<<endl;
  }
  cout<<"\n_______________________________________"<<endl;
}
