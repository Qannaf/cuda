
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>
#include <chrono>

using namespace std;

const int M = 1 << 3;          //8
const int N = 1 << 3;
const int K = 1 << 3;
const int SHMEM_SIZE = 1 << 3;   //4

__global__ void matrixMul(const int *a, const int *b, int *c);
void verify_result(vector<int> &a, vector<int> &b, vector<int> &c);
void afficheMatrix(vector<int>& m,int line, int colone);
auto get_time() { return chrono::high_resolution_clock::now(); }

int main() 
{
  size_t bytes_a = M * K * sizeof(int);
  size_t bytes_b = K * N * sizeof(int);
  size_t bytes_c = M * N * sizeof(int);

  // CPU
  vector<int> h_a(M * K);
  vector<int> h_b(K * N);
  vector<int> h_c(M * N);
  generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
  generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

  // GPU
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes_a);
  hipMalloc(&d_b, bytes_b);
  hipMalloc(&d_c, bytes_c);

  // CPU ---> GPU 
  hipMemcpy(d_a, h_a.data(), bytes_a, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), bytes_b, hipMemcpyHostToDevice);

  
  int THREADS = 1 << 1;              //2^1 =2
  int BLOCKS_X = N / THREADS;        //4
  int BLOCKS_Y = M / THREADS;        //4

  dim3 threads(THREADS, THREADS);
  dim3 blocks(BLOCKS_X, BLOCKS_Y);

  auto start = get_time();
  matrixMul<<<blocks, threads>>>(d_a, d_b, d_c);          // <<< (2,2),(4,4) >>>
  hipMemcpy(h_c.data(), d_c, bytes_c, hipMemcpyDeviceToHost);
 
  auto finish = get_time();
  auto duration =
      chrono::duration_cast<chrono::milliseconds>(finish - start);
    
  cout << "temps écoulé en kernel = " << duration.count() << " ms\n";
  afficheMatrix(h_a,M,N);
  afficheMatrix(h_b,M,N);
  afficheMatrix(h_c,M,N);

  


  verify_result(h_a, h_b, h_c);

  cout << "terminé avec succès"<<endl;


  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}





__global__ void matrixMul(const int *a, const int *b, int *c) 
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ int s_a[SHMEM_SIZE];
  __shared__ int s_b[SHMEM_SIZE];

  int tmp = 0;

  
  for (int i = 0; i < K; i += blockDim.x) 
  {
    s_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * K + i + threadIdx.x];
    s_b[threadIdx.y * blockDim.x + threadIdx.x] = b[i * N + threadIdx.y * N + col];
    __syncthreads();

    for (int j = 0; j < blockDim.x; j++) 
      tmp += s_a[threadIdx.y * blockDim.x + j] * s_b[j * blockDim.x + threadIdx.x];
    __syncthreads();
  }

  c[row * N + col] = tmp;
}


void verify_result(vector<int> &a, vector<int> &b, vector<int> &c) 
{
  for (int row = 0; row < M; row++) 
    for (int col = 0; col < N; col++) 
    {
      int tmp = 0;
      for (int i = 0; i < K; i++) 
        tmp += a[row * K + i] * b[i * N + col];
    
      assert(tmp == c[row * N + col]);
    }

}


void afficheMatrix(vector<int>& m,int line, int colone)
{ 
  for (int i = 0; i <line;  i++) 
  {
    for (int j = 0; j < colone; j++)
    { 
      cout<<m[i]<<" ";
    }
  cout<<endl;
  }
  cout<<"\n_______________________________________"<<endl;
}
