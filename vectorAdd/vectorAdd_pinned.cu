
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <vector>
#include <chrono>
#include <random>

using namespace std;
//===========================       kernel    ========================================
__global__ void vectorAdd(int *a, int *b, int *c, int N) {
  int i = blockIdx.x*blockDim.x + threadIdx.x; 
	int j = blockIdx.y*blockDim.y + threadIdx.y; 
	int k = j*gridDim.x * blockDim.x + i;  

	if (k < N) c[k] = a[k] + b[k]; 
}

auto get_time() { return chrono::high_resolution_clock::now(); }



//===========================       fuction main      ===================================================
int main() {
  constexpr int N = 1000 << 16;
  size_t bytes = sizeof(int) * N;
  int NUM_THREADS = 1 << 10;
  int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

  // CPU
  int *h_a, *h_b, *h_c;
  hipHostMalloc(&h_a, bytes, hipHostMallocDefault);
  hipHostMalloc(&h_b, bytes, hipHostMallocDefault);
  hipHostMalloc(&h_c, bytes, hipHostMallocDefault);

  for (int i = 0; i < N; i++) // initialisation les vacteurs a ,b
  {
    h_a[i]=rand() % 100;
    h_b[i]=rand() % 100;
  }
  
  
  //GPU
  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  // CPU ---> GPU
  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);


  auto start = get_time();

  vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

  // GPU ---> CPU
  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
  auto finish = get_time();
  auto duration =
      chrono::duration_cast<std::chrono::milliseconds>(finish - start);

  cout << "temps écoulé en kernel = " << duration.count() << " ms\n";

  for (int i = 0; i < N; i++) {
    assert(h_c[i] == h_a[i] + h_b[i]);
  }


  hipHostFree(h_a);
  hipHostFree(h_b);
  hipHostFree(h_c);
 
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  cout << "terminé avec succès"<<endl;

  return 0;
}
