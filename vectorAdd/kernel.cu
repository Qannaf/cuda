#include "hip/hip_runtime.h"
﻿/*
#include "hip/hip_runtime.h"
#include ""

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <vector>
#include <chrono>
#include <random>

using namespace std;
//===========================       kernel    ========================================
__global__ void vectorAdd(int* a, int* b, int* c, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = j * gridDim.x * blockDim.x + i;

    if (k < N) c[k] = a[k] + b[k];
}

auto get_time() { return chrono::high_resolution_clock::now(); }



//===========================       fuction main      ===================================================
int main() {
    constexpr int N = 1000 << 16;
    size_t bytes = sizeof(int) * N;
    int NUM_THREADS = 1 << 10;
    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

    // CPU
    int* h_a, * h_b, * h_c;
    hipHostMalloc(&h_a, bytes);
    hipHostMalloc(&h_b, bytes);
    hipHostMalloc(&h_c, bytes);

    for (int i = 0; i < N; i++) // initialisation les vacteurs a ,b
    {
        h_a[i] = rand() % 100;
        h_b[i] = rand() % 100;
    }


    //GPU
    int* d_a, * d_b, * d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // CPU ---> GPU
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);


    auto start = get_time();

    vectorAdd << <NUM_BLOCKS, NUM_THREADS >> > (d_a, d_b, d_c, N);

    // GPU ---> CPU
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    auto finish = get_time();
    auto duration =
        chrono::duration_cast<std::chrono::milliseconds>(finish - start);

    cout << "temps ecoule en kernel = " << duration.count() << " ms\n";

    for (int i = 0; i < N; i++) {
        assert(h_c[i] == h_a[i] + h_b[i]);
    }


    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    cout << "terminé avec succès" << endl;

    return 0;
}
*/

#include "hip/hip_runtime.h"
#include ""

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <iterator>
#include <vector>
#include <chrono>
#include <random>
#include <cstdio>
#include <cstdlib>

#define gpuErrchk(ans) { gpuAssert( (ans), __FILE__, __LINE__ ); }

inline void
gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (hipSuccess != code)
    {
        fprintf(stderr, "\nGPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }

    return;

} /* gpuAssert */

__global__ void Add(int N, int Offset, float* devA, float* devB, float* devC)
{

    for (int idx = blockIdx.x * blockDim.x + threadIdx.x + Offset; idx < N; idx += blockDim.x * gridDim.x)

        devC[idx] = devA[idx] + devB[idx];

}

int main()
{

    int N = 400000000;

    int Threads = 256;

    const int NbStreams = 8;

    float* A, * B, * C;
    gpuErrchk(hipHostAlloc((void**)&A, N * sizeof(*A), hipHostMallocDefault));
    gpuErrchk(hipHostAlloc((void**)&B, N * sizeof(*B), hipHostMallocDefault));
    gpuErrchk(hipHostAlloc((void**)&C, N * sizeof(*C), hipHostMallocDefault));

    for (int i = 0; i < N; i++)
    {
        A[i] = i;
        B[i] = i + 1;
    }

    float* devA, * devB, * devC;
    gpuErrchk(hipMalloc((void**)&devA, N * sizeof(*devA)));
    gpuErrchk(hipMalloc((void**)&devB, N * sizeof(*devB)));
    gpuErrchk(hipMalloc((void**)&devC, N * sizeof(*devC)));

    hipEvent_t EventPre,
        EventPost;
    float PostPreTime;

    gpuErrchk(hipEventCreate(&EventPre));
    gpuErrchk(hipEventCreate(&EventPost));

    hipStream_t Stream[NbStreams];
    for (int i = 0; i < NbStreams; i++)
        gpuErrchk(hipStreamCreate(&Stream[i]));

#ifdef NOSTREAMS

    gpuErrchk(hipEventRecord(EventPre));

    gpuErrchk(hipMemcpy(devA, A, N * sizeof(*A), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(devB, B, N * sizeof(*B), hipMemcpyHostToDevice));
    //        gpuErrchk( hipMemcpy(devC, C, N * sizeof(*C), hipMemcpyHostToDevice) );

    Add << < N / Threads, Threads >> > (N, 0, devA, devB, devC);

    //        gpuErrchk( hipMemcpy(A, devA, N * sizeof(*A), hipMemcpyDeviceToHost) );
    //        gpuErrchk( hipMemcpy(B, devB, N * sizeof(*B), hipMemcpyDeviceToHost) );
    gpuErrchk(hipMemcpy(C, devC, N * sizeof(*C), hipMemcpyDeviceToHost));

    gpuErrchk(hipEventRecord(EventPost));
    gpuErrchk(hipEventSynchronize(EventPost));
    gpuErrchk(hipEventElapsedTime(&PostPreTime, EventPre, EventPost));
    printf("\nTime not using streams: %f ms\n", PostPreTime);

#else

    const int StreamSize = N / NbStreams;
    gpuErrchk(hipEventRecord(EventPre));
    for (int i = 0; i < NbStreams; i++)
    {
        int Offset = i * StreamSize;

        gpuErrchk(hipMemcpyAsync(&devA[Offset], &A[Offset], StreamSize * sizeof(*A), hipMemcpyHostToDevice, Stream[i]));
        gpuErrchk(hipMemcpyAsync(&devB[Offset], &B[Offset], StreamSize * sizeof(*B), hipMemcpyHostToDevice, Stream[i]));
        //                gpuErrchk( hipMemcpyAsync(&devC[ Offset ], &C[ Offset ], StreamSize * sizeof(*C), hipMemcpyHostToDevice, Stream[ i ]) );

        Add << < StreamSize / Threads, Threads, 0, Stream[i] >> > (Offset + StreamSize, Offset, devA, devB, devC);

        //                gpuErrchk( hipMemcpyAsync(&A[ Offset ], &devA[ Offset ], StreamSize * sizeof(*devA), hipMemcpyDeviceToHost, Stream[ i ]) );
        //                gpuErrchk( hipMemcpyAsync(&B[ Offset ], &devB[ Offset ], StreamSize * sizeof(*devB), hipMemcpyDeviceToHost, Stream[ i ]) );
        gpuErrchk(hipMemcpyAsync(&C[Offset], &devC[Offset], StreamSize * sizeof(*devC), hipMemcpyDeviceToHost, Stream[i]));

    }

    gpuErrchk(hipEventRecord(EventPost));
    gpuErrchk(hipEventSynchronize(EventPost));
    gpuErrchk(hipEventElapsedTime(&PostPreTime, EventPre, EventPost));
    printf("\nTime using streams: %f ms\n", PostPreTime);

#endif /* ! USE_STREAMS */

    for (int i = 0; i < N; i++)
        if (C[i] != (A[i] + B[i])) { printf("mismatch at %d, was: %f, should be: %f\n", i, C[i], (A[i] + B[i])); return 1; }

    for (int i = 0; i < NbStreams; i++)
        gpuErrchk(hipStreamDestroy(Stream[i]));

    gpuErrchk(hipFree(devA));
    gpuErrchk(hipFree(devB));
    gpuErrchk(hipFree(devC));

    gpuErrchk(hipHostFree(A));
    gpuErrchk(hipHostFree(B));
    gpuErrchk(hipHostFree(C));

    gpuErrchk(hipEventDestroy(EventPre));
    gpuErrchk(hipEventDestroy(EventPost));

    printf("\n");

    return 0;

}