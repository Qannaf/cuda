#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include <iostream>
#include <chrono>
#include <random>

using namespace std;

//===========================       prototypes des fonctions    ========================================
__global__ void vectorAdd(int *a, int *b, int *c, int N) {
  int i = blockIdx.x*blockDim.x + threadIdx.x; 
	int j = blockIdx.y*blockDim.y + threadIdx.y; 
	int k = j*gridDim.x * blockDim.x + i;  

	if (k < N) c[k] = a[k] + b[k]; 
}

auto get_time() { return chrono::high_resolution_clock::now(); }





//===========================       fuction main      ===================================================
int main() {
  
  const int N = 1000 << 16;
  size_t bytes = N * sizeof(int);
  int BLOCK_SIZE = 1 << 10;
  int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
  

  int *a, *b, *c;
  int id = hipGetDevice(&id);

  hipMallocManaged(&a, bytes);
  hipMallocManaged(&b, bytes);
  hipMallocManaged(&c, bytes);
  
  
  

  hipMemAdvise(a, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemAdvise(b, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
  hipMemPrefetchAsync(c, bytes, id);

  for (int i = 0; i < N; i++) {
    a[i] = rand() % 100;
    b[i] = rand() % 100;
  }
  
  hipMemAdvise(a, bytes, hipMemAdviseSetReadMostly, id);
  hipMemAdvise(b, bytes, hipMemAdviseSetReadMostly, id);
  hipMemPrefetchAsync(a, bytes, id);
  hipMemPrefetchAsync(b, bytes, id);

  auto start = get_time();
  vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c, N);
  hipDeviceSynchronize();

  hipMemPrefetchAsync(a, bytes, hipCpuDeviceId);
  hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);
  hipMemPrefetchAsync(c, bytes, hipCpuDeviceId);

  auto finish = get_time();
  auto duration =
      chrono::duration_cast<std::chrono::milliseconds>(finish - start);

  cout << "temps écoulé en kernel = " << duration.count() << " ms\n";
  
  for (int i = 0; i < N; i++) {
    assert(c[i] == a[i] + b[i]);
  }

  
  hipFree(a);
  hipFree(b);
  hipFree(c);

  cout << "terminé avec succès"<<endl;


  return 0;
}
