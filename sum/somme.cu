// File name: add.cu

#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void add(int a, int b, int *c){
 *c = a+b;
}
int main(void){
 int c;
 int *device_c;
 hipMalloc((void**)&device_c, sizeof(int));
 add<<<1, 1>>>(2, 7, device_c);
 hipMemcpy(&c, device_c, sizeof(int), hipMemcpyDeviceToHost);
 printf("2+7 = %d\n", c);
 //cudeFree(device_c);
 return 0;
}