#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdio.h>
#include "hip/hip_runtime.h"

__global__ void Sum( int* index)

{
	int row = blockIdx.z * blockDim.z + threadIdx.z;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int depth = blockIdx.x * blockDim.x + threadIdx.x;
	
	atomicAdd(index, 1);	
}

int main(int argc, char* argv[])

{
	int W = 16*16*16;
	int H = 4*4*4;

	int* d_index = 0;
	int h_index = 0;

	hipMalloc((void**)&d_index, sizeof(int));
	hipMemcpy(d_index, &h_index, sizeof(int), hipMemcpyHostToDevice);
	dim3 grid = dim3(16, 16, 16);
	dim3 block = dim3(4, 4, 4);
	Sum << <grid,block>> > ( d_index);
	hipMemcpy(&h_index, d_index, sizeof(int), hipMemcpyDeviceToHost);
	fprintf(stderr, "%d\t %d\n", h_index,W*H);
	hipFree(d_index);

	return 0;

}