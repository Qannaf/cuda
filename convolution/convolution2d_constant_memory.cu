
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <chrono>

using namespace std;
#define MASK_DIM 7
#define MASK_OFFSET (MASK_DIM / 2)

// allocation in constant memory
__constant__ int mask[7 * 7];

__global__ void convolution_2d(int *matrix, int *result, int N);
void verify_result(int *m, int *mask, int *result, int N) ;
void init_matrix(int *m, int n);
auto get_time() { return chrono::high_resolution_clock::now(); }


int main() 
{
  int N = 1 << 10;
  size_t bytes_n = N * N * sizeof(int);
  size_t bytes_m = MASK_DIM * MASK_DIM * sizeof(int);

  // CPU
  int *matrix = new int[N * N];
  int *result = new int[N * N];
  int *h_mask = new int[MASK_DIM * MASK_DIM];

  init_matrix(matrix, N);
  init_matrix(h_mask, MASK_DIM);
  

  // GPU
  int *d_matrix;
  int *d_result;
  hipMalloc(&d_matrix, bytes_n);
  hipMalloc(&d_result, bytes_n);

  // CPU ---> GPU
  hipMemcpy(d_matrix, matrix, bytes_n, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

  
  int THREADS = 16;
  int BLOCKS = (N + THREADS - 1) / THREADS;
  dim3 block_dim(THREADS, THREADS);
  dim3 grid_dim(BLOCKS, BLOCKS);
  auto start = get_time();
  convolution_2d<<<grid_dim, block_dim>>>(d_matrix, d_result, N);

  // GPU --> CPU
  hipMemcpy(result, d_result, bytes_n, hipMemcpyDeviceToHost);
  auto finish = get_time();
  auto duration =
      chrono::duration_cast<std::chrono::milliseconds>(finish - start);
  
  cout << "temps écoulé en kernel = " << duration.count() << " ms\n";
  // Fonction test
  verify_result(matrix, h_mask, result, N);

  cout << "terminé avec succès"<<endl;

  
  hipFree(d_matrix);
  hipFree(d_result);
  delete[] matrix;
  delete[] result;
  delete[] h_mask;


  return 0;
}











__global__ void convolution_2d(int *matrix, int *result, int N) 
{
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  int start_r = row - MASK_OFFSET;
  int start_c = col - MASK_OFFSET;
  
  int temp = 0;

  for (int i = 0; i < MASK_DIM; i++) 
    for (int j = 0; j < MASK_DIM; j++) 
      if ((start_r + i) >= 0 && (start_r + i) < N) 
        if ((start_c + j) >= 0 && (start_c + j) < N) 
          temp += matrix[(start_r + i) * N + (start_c + j)] *mask[i * MASK_DIM + j];
    
  result[row * N + col] = temp;
}


void init_matrix(int *m, int n) 
{
  for (int i = 0; i < n; i++) 
    for (int j = 0; j < n; j++) 
      m[n * i + j] = rand() % 100;
}
    


void verify_result(int *m, int *mask, int *result, int N) 
{
  int temp;
  int offset_r,offset_c;

  for (int i = 0; i < N; i++) 
    for (int j = 0; j < N; j++) 
    {
      temp = 0;
      for (int k = 0; k < MASK_DIM; k++) 
      {
        offset_r = i - MASK_OFFSET + k;
        for (int l = 0; l < MASK_DIM; l++) 
        {
          offset_c = j - MASK_OFFSET + l;
          if (offset_r >= 0 && offset_r < N) 
            if (offset_c >= 0 && offset_c < N) 
              temp += m[offset_r * N + offset_c] * mask[k * MASK_DIM + l];
            
          
        }
      }
      assert(result[i * N + j] == temp);
    }
  
}